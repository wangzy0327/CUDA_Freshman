#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CUDA_CHECK_ERROR(call) \
{\
    const hipError_t error = call;\
    if (error != hipSuccess) { \
        const char* errorName = NULL;\
        hipDrvGetErrorName(error,&errorName);\
        const char* errorString = NULL;\
        hipDrvGetErrorString(error,&errorString);\
        printf("CUDA error Name : %s , line %d\n description : %s \n", errorName, __LINE__, errorString); \
        exit(error); \
    }\
}    

int main() {
    hipError_t err;

    // Initialize CUDA
    err = hipInit(0);
    CUDA_CHECK_ERROR(err);

    // Create CUDA context
    hipCtx_t cuContext;
    err = hipCtxCreate(&cuContext, 0, 0);
    CUDA_CHECK_ERROR(err);

    // Load module
    hipModule_t cuModule;
    err = hipModuleLoad(&cuModule, "kernel.ptx");
    CUDA_CHECK_ERROR(err);

    // Get kernel function
    hipFunction_t cuFunction;
    err = hipModuleGetFunction(&cuFunction, cuModule, "_Z15kernel_functionPfi");
    CUDA_CHECK_ERROR(err);

    // Allocate device memory
    hipDeviceptr_t d_data;
    int size = 256;
    err = hipMalloc(&d_data, size * sizeof(float));
    float *h_data = (float *)malloc(size*sizeof(float));
    for(int i = 0;i < size;i++)
        h_data[i] = i;
    hipMemcpyHtoD(d_data, h_data, size*sizeof(float));
    CUDA_CHECK_ERROR(err);

    // Set kernel parameters
    void *args[] = { &d_data, &size };
    err = hipModuleLaunchKernel(cuFunction, 1, 1, 1,  // grid dims
                         256, 1, 1,           // block dims
                         0, NULL,             // shared mem and stream
                         args, 0);            // kernel arguments
    CUDA_CHECK_ERROR(err);

    // Synchronize to wait for kernel completion
    err = hipCtxSynchronize();
    CUDA_CHECK_ERROR(err);

    // Cleanup
    hipFree(d_data);
    hipCtxDestroy(cuContext);

    return 0;
}
