#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipblas.h>
#include "freshman.h"
#include "mysgemms.cuh"
#define TILEX 32
#define TILEY 32

#define CEIL_DIV(m,n) ( (m) + (n) - 1 ) / (n)

void sgemm_CPU(int M, int N, int K, float alpha, const float * MatA, const float * MatB, float beta, float * MatC)
{   
    //列主序
    for(int i = 0;i < M;i++){
        for(int j = 0;j < N;j++){
            const float* A = MatA + i;
            const float* B = MatB + j*K;
            // float* C = MatC + j*M;
            float sum = 0.0f;
            for(int k = 0;k < K;k++){
                // sum += MatA[i+k*M] * MatB[k+j*K];
                sum += A[k*M] * B[k];
                // C[i] += A[k*M] * B[k];
            }
            MatC[i+j*M] = alpha * sum + beta * MatC[i+j*M];
        }
    }
}

void testThreadIdx(){
  int tx_size = 256;
  int bx = 0;int by = 0;
  for(int tx = 0;tx < tx_size;tx++){
    int row_b = tx%32, col_b = ((tx>>5)&7)<<1;  //16 x 64 row_b [0,1,2,...,31]  col_b [0,2,...,14]
    printf("kernel threadIdx : %d (%d,%d) =  (%d,%d),(%d,%d),(%d,%d),(%d,%d) transpose to (%d,%d),(%d,%d),(%d,%d),(%d,%d) \n", tx,\
    col_b,row_b,\
    row_b,(row_b%16+col_b)%KS_7_2,\
    row_b,(row_b%16+col_b+1)%KS_7_2,\
    row_b+32,(row_b%16+col_b)%KS_7_2,\
    row_b+32,(row_b%16+col_b+1)%KS_7_2,\
    (row_b%16+col_b)%KS_7_2,row_b,\
    (row_b%16+col_b+1)%KS_7_2,row_b,\
    (row_b%16+col_b)%KS_7_2,row_b+32,\
    (row_b%16+col_b+1)%KS_7_2,row_b+32);
  }
}

void testThreadIdx2(){
  int tx_size = 256;
  for(int tx = 0;tx < tx_size;tx++){
    int row = tx&0x1F;  // 0...31
    int col0,col1,col2,col3;
    col0 = (tx>>5)*4;  // col0 ∈ {0,4,8,12,16,20,24,28}
    col1 = col0 + 1;
    col2 = col0 + 2;
    col3 = col0 + 3;
    printf("threadIdx2 %d = transpose to (%d,%d),(%d,%d),(%d,%d),(%d,%d) \n",tx,(row+col0)%NS,row,(row+col1)%NS,row,(row+col2)%NS,row,(row+col3)%NS,row);
  }
}

void test_mysgemm_v1(int M, int N, int K, float alpha, const float* A, const float* B, float beta, float* C){
    // hipDeviceSynchronize();
    int blockX = 32, blockY = 32;
    dim3 blockDim(blockX,blockY);
    dim3 gridDim(CEIL_DIV(M,blockX),CEIL_DIV(N,blockY));
    mysgemm_v1<<<gridDim, blockDim>>>(M,N,K,alpha,A,B,beta,C);
    hipDeviceSynchronize();
    // hipStreamSynchronize(0);
}

void test_mysgemm_v2(int M, int N, int K, float alpha, const float* A, const float* B, float beta, float* C){
    // hipDeviceSynchronize();
    int blockX = 32, blockY = 32;
    dim3 blockDim(blockX,blockY);
    dim3 gridDim(CEIL_DIV(M,blockX),CEIL_DIV(N,blockY));
    mysgemm_v2<<<gridDim, blockDim>>>(M,N,K,alpha,A,B,beta,C);
    hipDeviceSynchronize();
}

void test_mysgemm_v3(int M, int N, int K, float alpha, const float* A, const float* B, float beta, float* C){
    // hipDeviceSynchronize();
    int blockX = 32, blockY = 32;
    dim3 blockDim(blockX,blockY);
    dim3 gridDim(CEIL_DIV(M,blockX),CEIL_DIV(N,blockY));
    mysgemm_v3<<<gridDim, blockDim>>>(M,N,K,alpha,A,B,beta,C);
    hipDeviceSynchronize();
}

void test_mysgemm_v4(int M, int N, int K, float alpha, const float* A, const float* B, float beta, float* C){
    // hipDeviceSynchronize();
    int blockX = 32, blockY = 32;
    dim3 blockDim(blockX,blockY);
    dim3 gridDim(CEIL_DIV(M,blockX),CEIL_DIV(N,blockY));
    mysgemm_v4<<<gridDim, blockDim>>>(M,N,K,alpha,A,B,beta,C);
    hipDeviceSynchronize();
}

void test_mysgemm_v5(int M, int N, int K, float alpha, const float* A, const float* B, float beta, float* C){
    // hipDeviceSynchronize();
    int blockX = 32, blockY = 32;
    // dim3 blockDim(1024);
    dim3 blockDim(256);//x4
    // dim3 blockDim(64);//x4
    dim3 gridDim(CEIL_DIV(M,blockX),CEIL_DIV(N,blockY));
    mysgemm_v5_ano2_pro<<<gridDim, blockDim>>>(M,N,K,alpha,A,B,beta,C);
    hipDeviceSynchronize();
}

void test_mysgemm_v6(int M, int N, int K, float alpha, const float* A, const float* B, float beta, float* C){
    // hipDeviceSynchronize();
    int blockX = 32, blockY = 32;
    // dim3 blockDim(1024);
    dim3 blockDim(256);//x4
    // dim3 blockDim(64);//x4
    dim3 gridDim(CEIL_DIV(M,blockX),CEIL_DIV(N,blockY));
    mysgemm_v6<<<gridDim, blockDim>>>(M,N,K,alpha,A,B,beta,C);
    hipDeviceSynchronize();
}

void test_mysgemm_v7(int M, int N, int K, float alpha, const float* A, const float* B, float beta, float* C){
    hipDeviceSynchronize();
    int blockX = 64, blockY = 64;
    // dim3 blockDim(1024);
    dim3 blockDim(256);//x4
    // dim3 blockDim(64);//x4
    dim3 gridDim(CEIL_DIV(M,blockX),CEIL_DIV(N,blockY));
    mysgemm_v7_ano_plus<<<gridDim, blockDim>>>(M,N,K,alpha,A,B,beta,C);
    hipDeviceSynchronize();
}


int main(int argc,char **argv)
{
  // set up device
  initDevice(0);
  int kernel=1;
  if(argc>=2)
    kernel=atoi(argv[1]);
  int SIZE[24];
  for (int i=0;i<24;i++) SIZE[i]=(i+1)<<8;
  if (kernel<0||kernel>11) {
    printf("Please enter a valid kernel number (0-11).\n");
    exit(-2);
  }
  // testThreadIdx();
  int m, n, k,max_size;
  int N=1, upper_limit;
  if (kernel<=7&&kernel!=0) upper_limit=8;
  else upper_limit=(sizeof(SIZE)/sizeof(int));
  max_size=SIZE[upper_limit-1];
  float* A_host = NULL,*B_host = NULL, *C_host = NULL,*C_from_dev = NULL,*C_from_dev_lib = NULL;
  float* A_dev = NULL,*B_dev = NULL,*C_dev = NULL,*C_dev_lib = NULL;
  float alpha = 1.0, beta = 0.;//two arbitary input parameters

  int nElem = max_size*max_size;
  int nBytes = sizeof(float)*max_size*max_size;

  CHECK(hipHostAlloc((float**)&A_host,nBytes,hipHostMallocDefault));
  CHECK(hipHostAlloc((float**)&B_host,nBytes,hipHostMallocDefault));
  CHECK(hipHostAlloc((float**)&C_host,nBytes,hipHostMallocDefault));
  CHECK(hipHostAlloc((float**)&C_from_dev,nBytes,hipHostMallocDefault));
  CHECK(hipHostAlloc((float**)&C_from_dev_lib,nBytes,hipHostMallocDefault));
  //Malloc
  // A_host=(float*)malloc(nBytes);
  // B_host=(float*)malloc(nBytes);
  // C_host=(float*)malloc(nBytes);
  // C_from_dev=(float*)malloc(nBytes);
  // C_from_dev_lib=(float*)malloc(nBytes);

  CHECK(hipMalloc((float**)&A_dev,nBytes));
  CHECK(hipMalloc((float**)&B_dev,nBytes));
  CHECK(hipMalloc((float**)&C_dev,nBytes));
  CHECK(hipMalloc((float**)&C_dev_lib,nBytes));

  memset(C_host,0,nBytes);
  memset(C_from_dev,0,nBytes);
  memset(C_from_dev_lib,0,nBytes);
  CHECK(hipMemset(C_dev,0,nBytes));
  CHECK(hipMemset(C_dev_lib,0,nBytes));

  initialData(A_host,nElem);
  initialData(B_host,nElem);

  CHECK(hipMemcpyAsync(A_dev,A_host,nBytes,hipMemcpyHostToDevice,0));
  CHECK(hipMemcpyAsync(B_dev,B_host,nBytes,hipMemcpyHostToDevice,0));
  // CHECK(hipMemcpy(A_dev,A_host,nBytes,hipMemcpyHostToDevice));
  // CHECK(hipMemcpy(B_dev,B_host,nBytes,hipMemcpyHostToDevice));

  hipSharedMemConfig MemConfig = hipSharedMemBankSizeFourByte;
  hipblasHandle_t handle; hipblasCreate(&handle);
  hipEvent_t beg_lib, end_lib, beg, end;
  hipEventCreate(&beg); hipEventCreate(&end);
  hipEventCreate(&beg_lib); hipEventCreate(&end_lib);
  float elapsed_time;
  
  printf("--------------------------------------------\n");
  

  // for(int i_count = upper_limit-1;i_count < upper_limit;i_count++){
  // for(int i_count = 0;i_count < upper_limit;i_count++){
  for(int i_count = 0;i_count < 1;i_count++){
    m=n=k=SIZE[i_count];
    printf("\nM=N=K=%d:\n",m);
    //warmup cuBLAS 库在第一次调用时需要初始化内部状态（如加载内核、分配内部缓冲区等），这会带来额外的开销
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, A_dev, m, B_dev, k, &beta, C_dev_lib, m);
    hipEventRecord(beg_lib);
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, A_dev, m, B_dev, k, &beta, C_dev_lib, m);
    hipEventRecord(end_lib);
    hipEventSynchronize(beg_lib);
    hipEventSynchronize(end_lib);
    hipEventElapsedTime(&elapsed_time, beg_lib, end_lib);
    elapsed_time /= 1000.;
    printf("GPU cublas Average elasped time: %f second, performance: %f GFLOPS.\n", elapsed_time,2.*1e-9*m*n*k/elapsed_time);
    if(i_count < 0){
      //数据量低于 3*256 时，计算CPU校对结果
      double iStart=cpuSecond();
      sgemm_CPU(m, n, k, alpha, A_host, B_host, beta, C_host);
      double iElaps=cpuSecond()-iStart;
      printf("CPU Execution Time elapsed %f sec\n",iElaps);
    }
    CHECK(hipMemcpyAsync(C_from_dev_lib, C_dev_lib, nBytes, hipMemcpyDeviceToHost,0));
    // CHECK(hipMemcpy(C_from_dev_lib, C_dev_lib, nBytes, hipMemcpyDeviceToHost));
    hipDeviceSynchronize();
    elapsed_time = 0.0f;
    hipEventRecord(beg);
    for(int n_count = 0;n_count < N;n_count++){
      switch (kernel)
      {
        case 0: hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, A_dev, m, B_dev, k, &beta, C_dev, m);break;
        case 1: test_mysgemm_v1(m,n,k,alpha,A_dev,B_dev,beta,C_dev);break;
        case 2: test_mysgemm_v2(m,n,k,alpha,A_dev,B_dev,beta,C_dev);break;
        case 3: test_mysgemm_v3(m,n,k,alpha,A_dev,B_dev,beta,C_dev);break;
        case 4: test_mysgemm_v4(m,n,k,alpha,A_dev,B_dev,beta,C_dev);break;
        case 5: test_mysgemm_v5(m,n,k,alpha,A_dev,B_dev,beta,C_dev);break;
        case 6: test_mysgemm_v6(m,n,k,alpha,A_dev,B_dev,beta,C_dev);break;
        case 7: test_mysgemm_v7(m,n,k,alpha,A_dev,B_dev,beta,C_dev);break;
        default:
          break;
      }
    }
    hipEventRecord(end);
    hipEventSynchronize(beg);
    hipEventSynchronize(end);
    hipEventElapsedTime(&elapsed_time, beg, end);
    elapsed_time /= 1000.;
    printf("GPU mySgemm Average elasped time: %f second, performance: %f GFLOPS.\n", elapsed_time/N,2.*1e-9*N*m*n*k/elapsed_time);
    fflush(stdout);
    CHECK(hipMemcpyAsync(C_from_dev, C_dev, nBytes, hipMemcpyDeviceToHost,0));
    // CHECK(hipMemcpy(C_from_dev, C_dev, nBytes, hipMemcpyDeviceToHost));
    hipDeviceSynchronize();
    // if(i_count < 3)
    //   checkResult(C_host,C_from_dev_lib,nElem);
    // else
    checkResult(C_from_dev,C_from_dev_lib,nElem);
    // printMatrix(C_from_dev_lib,n,1);
  }

  hipFree(A_dev);
  hipFree(B_dev);
  hipFree(C_dev);
  hipFree(C_dev_lib);
  hipHostFree(A_host);
  hipHostFree(B_host);
  hipHostFree(C_host);
  hipHostFree(C_from_dev);
  hipHostFree(C_from_dev_lib);
  hipblasDestroy(handle);
  // testThreadIdx();
  // testThreadIdx2();
  return 0;
}